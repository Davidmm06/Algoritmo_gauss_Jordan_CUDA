#include<stdlib.h>
#include<stdio.h>
#include<time.h>
#include "common.h"
/***/
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<>
/***/

/**************************/
/***Gaauss Jordan en cpu***/
/**************************/
void GaussJordan_cpu(float* AB, int n, int m, float* X, float time2) {
	hipEvent_t timer1, timer2;
	hipEventCreate(&timer1);
	hipEventCreate(&timer2);
	hipEventRecord(timer1, 0);
	hipEventSynchronize(timer1);
	for (int poscol = 0; poscol < n; poscol++) {
		for (int idx = 0; idx < m; idx++) {
			int pospivot = (n + 2) * poscol;
			int posfinfila = (n + 1) * (poscol + 1);
			float piv = AB[pospivot];
			for (int j = pospivot; j < posfinfila; j++) {
				AB[j] = AB[j] / piv;
			}
			int posfactor = pospivot % (n + 1) + idx * (n + 1);
			if (posfactor != pospivot) {
				float factor = AB[posfactor];
				for (int j = pospivot; j < posfinfila; j++) {
					int posactualelim = j % (n + 1) + idx * (n + 1);
					AB[posactualelim] = -
						1 * factor * AB[j] + AB[posactualelim];
				}
			}
		}
		int posultimacol = (poscol + 1) * (n + 1) - 1;
		X[poscol] = AB[posultimacol];
	}
	hipEventRecord(timer2, 0);
	hipEventSynchronize(timer1);
	hipEventSynchronize(timer2);
	hipEventElapsedTime(&time2, timer1, timer2);
}
void Generador(int tama�o, float* hostPtrL) {
	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < tama�o; i++) {
		hostPtrL[i] = (int)(rand() & 0xFF);
	}
}
void enter() {
	char enter = 0;
	while (enter != '\r' && enter != '\n')
		enter = getchar();
}